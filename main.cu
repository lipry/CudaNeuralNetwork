#include <iostream>
#include "src/utils/Matrix.h"
#include "hipblas.h"
#include "src/utils/common.h"
#include "src/layers/LinearLayer.h"

using namespace std;

int main() {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // Y(m,n) = W(m,k) * A(k,n)
    int features = 4;
    int n_entries = 2;
    Matrix A = Matrix(features, n_entries);
    A.allocate();
    int count = 1;
    for(int r = 0; r<features; r++){
        for(int c=0; c<n_entries;c++){
            A[CMIDX(r, c, features)] = count;
            count++;
        }
    }
    A.cpyHostToDev();
    cout << "A:" << endl;
    cout << A << endl;


    Layer *l = new LinearLayer("prova", 3, features); // W

    Matrix Y = l->forward(handle, A);

    Y.cpyDevToHost();
    cout << "Y: " << endl;
    cout<< Y << endl;


    hipDeviceReset();
    return 0;
}