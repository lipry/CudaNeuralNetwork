#include <iostream>
#include <cstring>
#include "src/utils/Matrix.h"
#include "hipblas.h"
#include "src/utils/common.h"
#include "src/layers/LinearLayer.h"
#include "src/layers/SigmoidLayer.h"
#include "src/cost_functions/BinaryCrossEntropy.h"
#include "src/NeuralNetwork.h"
#include "src/layers/ReluLayer.h"
#include "src/datasets/MNISTParser.h"

using namespace std;

int main() {

    std::string image_file = "/home/studenti/fabio.lipreri/Documents/NeuralNetworkCUDA/data/t10k-images-idx3-ubyte";
    std::string labels_file = "/home/studenti/fabio.lipreri/Documents/NeuralNetworkCUDA/data/t10k-labels-idx1-ubyte";

    MNISTDataset mnist;

    mnist.Parse(image_file.c_str(), labels_file.c_str());

    //cout << mnist.GetImageCount() << endl;
    //mnist.Print();

    //FILE* fimg = nullptr;
    //cout << fopen("/home/studenti/fabio.lipreri/Documents/NeuralNetworkCUDA/data/prova.txt", "r") << endl;
    //printf("Oh dear, something went wrong with read()! %s\n", strerror(errno));

    /*hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    int features = 2;
    int n_entries = 4;
    Matrix A = Matrix(features, n_entries);
    A.allocate();
    //int count = 1;
    for(int r = 0; r<features; r++){
        for(int c=0; c<n_entries;c++){
            A[CMIDX(r, c, features)] = 0;
            //count++;
        }
    }
    A[CMIDX(1, 0, features)] = -3;
    A[CMIDX(0, 3, features)] = 1;

    A.cpyHostToDev();

    cout << "A: " << endl;
    cout << A << endl;

    Matrix Y_Labels = Matrix(n_entries, 1);
    Y_Labels.allocate();
    for(int i = 0; i < n_entries; i++){
        Y_Labels[i] = 0.7;
    }

    Y_Labels[1] = 0.3;
    Y_Labels[3] = 0.9;

    Y_Labels.cpyHostToDev();


    NeuralNetwork nn = NeuralNetwork(1.0f);
    nn.newLayer(new LinearLayer("linear_layer1", 3, features));
    nn.newLayer(new ReluLayer("relu1"));
    nn.newLayer(new LinearLayer("linear_layer2", 1, 3));
    nn.newLayer(new SigmoidLayer("sigmoid_out"));

    nn.setCostFunction(new BinaryCrossEntropy());

    Matrix Y;
    for (int e = 0; e < 2; e ++) {
        cout << "EPOCA" << e << endl;
        Y = nn.forward(handle, A);


        Y.cpyDevToHost();
        cout << "Y: " << endl;
        cout << Y << endl;

        nn.backprop(handle, Y, Y_Labels);
    }*/


    // Y(m,n) = W(m,k) * A(k,n)
    /*
    cout << "A:" << endl;
    cout << A << endl;
    Matrix top_diff = Matrix(n_entries, features);
    top_diff.allocate();
    count = 0;
    for(int r = 0; r<n_entries; r++){
        for(int c=0; c<features; c++){
            top_diff[CMIDX(r, c, n_entries)] = count;
            count++;
        }
    }

    top_diff.cpyHostToDev();
    cout << "top_diff:" << endl;
    cout << top_diff << endl;

    SigmoidLayer *l = new SigmoidLayer("Sigmoid");
    //Layer *l = new LinearLayer("prova", 3, features); // W

    Matrix Y = l->forward(handle, A);

    Y.cpyDevToHost();
    cout << "Y: " << endl;
    cout<< Y << endl;

    Matrix b = l->backward(handle, top_diff, 1.0f);

    b.cpyDevToHost();
    cout << "sigmoid backward with mul" << endl;
    cout << b << endl;*/

    /*Matrix y = Matrix(5, 1);
    y.allocate();

    for(int i = 0; i  < y.getX(); i++){
        y[i] = 0.5;
    }

    y[2] = 0.5;
    y[4] = 0.2;


    Matrix labels = Matrix(5, 1);
    labels.allocate();

    for(int i = 0; i  < labels.getX(); i++){
        labels[i] = 0.5;
    }

    labels[2] = 0.1;
    labels[4] = 0.8;

    y.cpyHostToDev();
    labels.cpyHostToDev();

    Matrix dY = Matrix(5, 1);
    dY.allocate();

    BinaryCrossEntropy bce = BinaryCrossEntropy();
    float cost = bce.getCost(y, labels);

    //dY.cpyDevToHost();
    cout << "cost: " << cost << endl;*/

    hipDeviceReset();
    return 0;
}


