#include <iostream>
#include "src/utils/Matrix.h"
#include "hipblas.h"
#include "src/utils/common.h"
#include "src/layers/LinearLayer.h"
#include "src/layers/SigmoidLayer.h"
#include "src/CostFunctions/BinaryCrossEntropy.h"
#include "src/NeuralNetwork.h"

using namespace std;


int main() {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    int features = 2;
    int n_entries = 4;
    Matrix A = Matrix(features, n_entries);
    A.allocate();
    //int count = 1;
    for(int r = 0; r<features; r++){
        for(int c=0; c<n_entries;c++){
            A[CMIDX(r, c, features)] = 0;
            //count++;
        }
    }
    A[CMIDX(1, 0, features)] = 1;
    A[CMIDX(0, 3, features)] = 1;

    A.cpyHostToDev();

    cout << "A: " << endl;
    cout << A << endl;

    NeuralNetwork nn = NeuralNetwork(1.0f);
    nn.newLayer(new LinearLayer("linear_layer1", 3, features));
    nn.newLayer(new SigmoidLayer("sigmoid1"));
    nn.newLayer(new LinearLayer("linear_layer2", 1, features));
    nn.newLayer(new SigmoidLayer("sigmoid1"));

    Matrix Y = nn.forward(handle, A);

    Y.cpyDevToHost();

    cout << "Y: " << endl;
    cout << Y << endl;


    // Y(m,n) = W(m,k) * A(k,n)
    /*
    cout << "A:" << endl;
    cout << A << endl;
    Matrix top_diff = Matrix(n_entries, features);
    top_diff.allocate();
    count = 0;
    for(int r = 0; r<n_entries; r++){
        for(int c=0; c<features; c++){
            top_diff[CMIDX(r, c, n_entries)] = count;
            count++;
        }
    }

    top_diff.cpyHostToDev();
    cout << "top_diff:" << endl;
    cout << top_diff << endl;

    SigmoidLayer *l = new SigmoidLayer("Sigmoid");
    //Layer *l = new LinearLayer("prova", 3, features); // W

    Matrix Y = l->forward(handle, A);

    Y.cpyDevToHost();
    cout << "Y: " << endl;
    cout<< Y << endl;

    Matrix b = l->backward(handle, top_diff, 1.0f);

    b.cpyDevToHost();
    cout << "sigmoid backward with mul" << endl;
    cout << b << endl;*/

    /*Matrix y = Matrix(5, 1);
    y.allocate();

    for(int i = 0; i  < y.getX(); i++){
        y[i] = 0.5;
    }

    y[2] = 0.5;
    y[4] = 0.2;


    Matrix labels = Matrix(5, 1);
    labels.allocate();

    for(int i = 0; i  < labels.getX(); i++){
        labels[i] = 0.5;
    }

    labels[2] = 0.1;
    labels[4] = 0.8;

    y.cpyHostToDev();
    labels.cpyHostToDev();

    Matrix dY = Matrix(5, 1);
    dY.allocate();

    BinaryCrossEntropy bce = BinaryCrossEntropy();
    float cost = bce.getCost(y, labels);

    //dY.cpyDevToHost();
    cout << "cost: " << cost << endl;*/

    hipDeviceReset();
    return 0;
}


