#include "hip/hip_runtime.h"
//
// Created by Fabio Lipreri on 2019-11-15.
//

#include "cudamath.h"
#include <stdio.h>
#include "hipblas.h"
#include "Matrix.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdexcept>

// ===============
// =   KERNELS   =
// ===============

/*__global__ void add_vect(float *R, float *A, float *B, int x, int y){
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < x*y)
        R[idx] = __fadd_rn(A[idx], B[idx]);
}*/

__global__ void init_randoms(unsigned int seed, hiprandState_t* states) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    hiprand_init(seed, index, 0, &states[index]);
}

__global__ void randoms(hiprandState_t* states, float* numbers, float lower, float higher) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    numbers[index] = lower + (higher - lower) * hiprand_uniform(&states[index]);
}

__global__ void add_vector_row_wise(float *R, float *A, float *V, int x, int y){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = CMIDX(row, col, x);
    if(row < x && col < y) {
        R[index] = A[index] + V[row];
    }
}

__device__ float sigmoid(float x){
    return __frcp_rn(__fadd_rn(1, exp(-x)));
}

__device__ float sigmoid_derivate(float x){
    return __fmul_rn(x, __fsub_rn(1.0f, x));
}

__global__ void sigmoidForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = sigmoid(V[index]);
}

__global__ void sigmoidBackward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index < x*y)
        R[index] = sigmoid_derivate(V[index]);
}

//TODO: rifare con parallel reduction
__global__ void binaryCrossEntropyCost(float* cost, float* predictions, float* target, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        float partial_cost = target[index] * logf(predictions[index])
                             + (1.0f - target[index]) * logf(1.0f - predictions[index]);
        atomicAdd(cost, -partial_cost / size);
    }
}


__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int x) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < x) {
        dY[index] = -1.0 * ( target[index]/predictions[index] - (1 - target[index])/(1 - predictions[index]) );
    }
}

// ========================
// =   KERNEL FUNCTIONS   =
// ========================

void gpu_add_bias(float *A, float *b, float *Y, int x, int y){
    dim3 TxB(BLOCK_SIZE, BLOCK_SIZE);
    dim3 num_blocks((x*y + TxB.x - 1) / TxB.x, (x*y + TxB.y - 1) / TxB.y);
    add_vector_row_wise<<<num_blocks, TxB>>>(Y, A, b, x, y);
}

void gpu_sigmoid_forward(float *Z, float *Res, int x, int y){
    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((x*y + TxB.x - 1) / TxB.x);
    sigmoidForward<<<num_blocks, TxB>>>(Res, Z, x, y);
}

void gpu_sigmoid_backward(float *Z, float *Res, int x, int y){
    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((x*y + TxB.x - 1) / TxB.x);
    sigmoidBackward<<<num_blocks, TxB>>>(Res, Z, x, y);
}

void gpu_bce_cost(float *cost, float *prediction, float *labels, int x){
    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((x + TxB.x - 1) / TxB.x);
    binaryCrossEntropyCost<<<num_blocks, TxB>>>(cost, prediction, labels, x);

    hipDeviceSynchronize(); // todo: serve?!
}

void gpu_derivative_bce_cost(float *dY, float* predictions, float* target, int x){
    dim3 TxB(BLOCK_SIZE);
    dim3 num_blocks((x + TxB.x - 1) / TxB.x);
    dBinaryCrossEntropyCost<<<num_blocks, TxB>>>(predictions, target, dY, x);
}

// ========================
// =   CUBLAS FUNCTIONS   =
// ========================

// Multiply the arrays A and B on GPU and save the result in C
// Y(m,n) = W(m,k) * A(k,n)
void gpu_blas_mmul(hipblasHandle_t &handle, const float *W, hipblasOperation_t W_op,
        const float *A, hipblasOperation_t A_op, float *Y,
        const int m, const int n, const int k, float learning_rate, const float batch_size, const float bet) {
    int lda = 0,ldb = 0,ldc = m;
    const float alf = (1.0f / batch_size) * learning_rate;
    const float *alpha = &alf;
    const float *beta = &bet;
    if(W_op == HIPBLAS_OP_N && A_op == HIPBLAS_OP_N) {
        lda = m;
        ldb = k;
    }else if (W_op == HIPBLAS_OP_T && A_op == HIPBLAS_OP_N){
        lda = k;
        ldb = k;
    }else if (W_op == HIPBLAS_OP_N && A_op == HIPBLAS_OP_T){
        lda = m;
        ldb = n;
    }else{
        throw std::invalid_argument("Operations not supported in gpu_blas_mmul");
    }

    hipblasSgemm(handle, W_op, A_op, m, n, k, alpha, W, lda, A, ldb, beta, Y, ldc);
}

// TODO: implementare eventualemente con parallel reduction
void gpu_blas_sum_column(hipblasHandle_t &handle, const float *W, float *Y, const int m, const int n, float learning_rate,
        const float batch_size, const float bet){
    int lda = m;
    const float alf = (1.0f / batch_size) * learning_rate;
    const float *alpha = &alf;
    const float *beta = &bet;

    //building a dummy 1s vector x
    Matrix x = Matrix(m, 1);
    x.allocate();
    for (int i = 0; i<m; i++)
        x[i] = 1.0f;
    x.cpyHostToDev();

    // Y = W * x
    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, alpha, W, lda, x.getDevData().get(), 1, beta, Y, 1);
}

/*void gpu_blas_mtmul(hipblasHandle_t &handle, const float *W, const float *A, float *Y, const int m,
                    const int n, const int k){
    int lda=k,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, alpha, W, lda, A, ldb, beta, Y, ldc);
}

void gpu_blas_mmtul(hipblasHandle_t &handle, const float *W, const float *A, float *Y, const int m, const int n,
                    const int k, const float batch_size, const float bet){
    int lda=m,ldb=n,ldc=m;
    const float *alpha = &batch_size;
    const float *beta = &bet;

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha, W, lda, A, ldb, beta, Y, ldc);
}*/


