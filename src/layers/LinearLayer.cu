#include <utility>
#include <bits/stdc++.h>

//
// Created by Fabio Lipreri on 2019-11-15.
//

#include "LinearLayer.h"
#include "../utils/common.h"
#include "../utils/cudamath.h"

using namespace std;

LinearLayer::LinearLayer(std::string name, size_t x, size_t y) :
W(x, y), b(x, 1)
{
    this->name = name;
    W.allocate();
    b.allocate();
    this->initWeights(true, 0.0f, 1.0f);
    this->initBias();

    W.cpyDevToHost();
    cout << "Weights" << endl;
    cout << W << endl;

    b.cpyDevToHost();
    cout << "BIAS" << endl;
    cout << b << endl;

}

LinearLayer::~LinearLayer()
{}

Matrix &LinearLayer::forward(hipblasHandle_t handle, Matrix &A) {
    this->A = A;

    Y.allocate_size(W.getX(), A.getY());

    // Y(m,n) = W(m,k) * A(k,n)
    gpu_blas_mmul(handle, this->W.getDevData().get(), this->A.getDevData().get(),
            this->Y.getDevData().get(), this->W.getX(), this->W.getY(), this->A.getY());

    gpu_add_bias(this->Y.getDevData().get(), this->b.getDevData().get(),
            this->Y.getDevData().get(), this->Y.getX(), this->Y.getY());

    return Y;
}

Matrix &LinearLayer::backward(hipblasHandle_t handle, Matrix &top_diff) {
    dA.allocate_size(A.getX(), A.getY());

    //k, m, n
    gpu_blas_mtmul(handle, this->W.getDevData().get(), top_diff.getDevData().get(),
                  this->dA.getDevData().get(), this->W.getX(), this->W.getY(), top_diff.getY());

    dA.cpyDevToHost();
    cout << dA << endl;
    return dA;
}

void LinearLayer::initWeights(bool random, float lower, float higher) {
    if(random){

        hiprandState_t* states;
        int N = this->W.getX()*this->W.getY();

        /* allocate space on the GPU for the random states */
        hipMalloc((void**) &states, N * sizeof(hiprandState_t));

        dim3 TxB(BLOCK_SIZE);
        dim3 num_blocks((N + TxB.x - 1) / TxB.x);
        init_randoms<<<num_blocks, TxB>>>(time(0), states);
        randoms<<<num_blocks, TxB>>>(states, this->W.getDevData().get(), lower, higher);

    }else{
        initZeroes(this->W, 3);
        this->W.cpyHostToDev();
    }

}

void LinearLayer::initBias() {
    this->initZeroes(this->b, 1);
    //this->b[0] = 1;
    //this->b[1] = 2;
    //this->b[2] = 3;
    this->b.cpyHostToDev();
}

void LinearLayer::initZeroes(Matrix &x, float n) {
    for(int r = 0; r<x.getX(); r++){
        for(int c = 0; c<x.getY(); c++){
            x[r*x.getY()+c] = n;
        }
    }
}


Matrix &LinearLayer::getWeigths() {
    return W;
}

Matrix &LinearLayer::getBias() {
    return b;
}
